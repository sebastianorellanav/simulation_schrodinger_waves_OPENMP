#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>       // for time()
#include "../incl/simulation.h"
#include "../incl/utils.h"

float C = 1.0; // Velocidad de la onda en el medio.
float DT = 0.1; // Intervalo de tiempo con que avanza la simulacion.
float DD = 2.0; // Cambio en la superficie.

__host__ int main(int argc, char *argv[]){

    int N = 0; // Tamano grilla.
    int x = 0; // NTamaño de bloque en x.
    int y = 0; // Tamaño de bloque en y.
    int T = 0; // Numero de pasos.
    const char* file = NULL; // Archivo de salida.

    int count; // Contador de opciones.
    int passcount = 0; // Contador de opciones validas.

    if (argc == 11){
        for( count = 0; count < argc; count++){

            if(strcmp( argv[count], "-N" ) == 0){
                sscanf(argv[count+1],"%d",&N);
                passcount +=1;
            }
            else if(strcmp( argv[count], "-x" ) == 0){
                sscanf(argv[count+1],"%d",&x);
                passcount +=1;
            }
            else if(strcmp( argv[count], "-y" ) == 0){
                sscanf(argv[count+1],"%d",&y);
                passcount +=1;
            }
            else if(strcmp( argv[count], "-f" ) == 0){
                file = argv[count+1];
                passcount +=1;
            }
            else if(strcmp( argv[count], "-T" ) == 0){
                sscanf(argv[count+1],"%d",&T);
                passcount +=1;
            }
        }
    }
    else{
        printf("¡No se han ingresado todas las opciones!.\n");
        return 404;
    }
    if(passcount == 5){    
        int i = 0;
        int j = 0;

        // Se asigna la memoria en CPU
        float *matrix_t0 = (float *)malloc(N*N*sizeof(float));
        float *matrix_t1 = (float *)malloc(N*N*sizeof(float));

        // Se inicializa la matriz 
        initialize_matrix(&matrix_t0, N);

        //Se inicializa el tamaño de grilla y de bloque
        dim3 blocksize;
        dim3 gridsize; 
        gridsize.x = N;
        gridsize.y = N;
        blocksize.x = x;
        blocksize.y = y;

        // Se asigna la memoria en GPU
        hipMalloc((void **) &m_t0, N*N*sizeof(float));
        hipMalloc((void **) &m_t1, N*N*sizeof(float));

        // Se copia la matriz de cpu a gpu
        hipMemcpy(m_t0, matrix_t0, N*N*sizeof(float), hipMemcpyHostToDevice);

        get_first_matrix<<<gridsize, blocksize>>>(float &m_t0, float &m_t1, int N);

        for(i = 0; i < N; i++){
            get_matrix<<<gridsize, blocksize>>>(float &m_t0, float &m_t1, int N);
        }

        hipMemcpy(matrix_t1, m_t1,  N*N*sizeof(float), hipMemcpyDeviceToHost);

        save_image(matrix_t1, N, file);


        return 1;
    }
    else{
        printf("¡Hay una o mas opciones invalidas!\n");
        return 404;
    }
}