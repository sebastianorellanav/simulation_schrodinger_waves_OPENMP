#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "../incl/constants.h"



__global__ void get_matrix(float **m1, float **m2,  int N){
    
    extern __shared__ float new[];
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int i = blockDim.y * blockIdx.y + threadIdx.y;

    new[i*N + j] = 0.0;

    float a1 = 0.0;
    float a2 = 0.0;
    float a3 = 0.0;
    float a4 = 0.0;  
    a1 = (*m1)[(i-1)*N +j];
    a2 = (*m1)[(i+1)*N + j];
    a3 = (*m1)[i*N + (j-1)];
    a4 = (*m1)[i*N + (j+1)];

    if(i > 0 && i < N-1 && j > 0 && j < N-1){
        new[i*N + j] = 2*(*m1)[i*N + j] - (*m2)[i*N + j] + ((C*C) * ((DT/DD)*(DT/DD)) * (a2+a1+a4+a3 - 4.0*(*m1)[i*N + j]));
    
    }

    (*m2)[i*N + j] = (*m1)[i*N + j];
    (*m1)[i*N + j] = new[i*N + j];

    hipFree(new);
    //return new;
}

__global__ void get_first_matrix(float **m_t0, float **m_t1, int N){

    
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    (*m_t1)[i*N +] = 0.0;

    float a1 = 0.0;
    float a2 = 0.0;
    float a3 = 0.0;
    float a4 = 0.0;  
    a1 = (*m)[(i-1)*N +j];
    a2 = (*m)[(i+1)*N + j];
    a3 = (*m)[i*N + (j-1)];
    a4 = (*m)[i*N + (j+1)];

    if(i > 0 && i < N-1 && j > 0 && j < N-1){
        (*m_t1)[i*N + j] = (*m)[i*N + j] + ((C*C) * ((DT*DT)/(DD*DD)) * (a2+a1+a4+a3 - 4.0*(*m)[i*N + j]));
    }
}



__host__ void initialize_matrix(float **matrix, int N){
    int i= 0;
    int j = 0;
    for(i = 0; i < N; i++){
        for(j = 0; j < N; j++){
            if((i > 0.4*N && i <0.6*N) && (j > 0.4*N && j < 0.6*N)){
                (*matrix)[i*N + j] = 20.0;
            }
            else{
                (*matrix)[i*N + j] = 0.0;
            }
        }
    }
}

